#include <iostream>
#include <chrono>
#include <cstdlib>
#include <random>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

typedef chrono::high_resolution_clock timer;

// check if there are any errors launching the kernel
#define cuda_error_check() { cuda_assert(__FILE__, __LINE__); }
inline void cuda_assert(const char *file, int line, bool abort = true)
{
    auto error = hipGetLastError();
    if (error != hipSuccess)
    {
        cerr << "CUDA error: "
             << hipGetErrorString(error)
             << " (" << error << ") -- "
             << file << " -- line: "
             << line << endl;
        if (abort) exit(error);
    }
}

// get input from the command line for total number of tosses
size_t process_cmdline(int argc, char* argv[])
{
    if (argc > 4)
    {
        cout << "Usage: "
             << argv[0]
             << " [number of tosses] [number of threads]"
             << endl;
        return 0;
    }
    else if (1 == argc)
        return 10'000'000;
    else
        return atoll(argv[1]);
}

// device kernel to perform Monte Carlo version of tossing darts at a board
__global__ void cuda_toss(size_t n, size_t* in)
{
    size_t rank = threadIdx.x;
    size_t size = blockDim.x;

    // Initialize RNG
    hiprandState_t rng;
    hiprand_init(clock64(), threadIdx.x + blockIdx.x * blockDim.x, 0, &rng);

    in[rank] = 0;                           // local number of points in circle
    for (size_t i = 0; i < n / size; ++i)
    {
        float x = hiprand_uniform(&rng);     // Random x position in [0,1]
        float y = hiprand_uniform(&rng);     // Random y position in [0,1]
        // if (x * x + y * y <= 1)          // is point in circle?
        //     ++in[rank];                  // increase thread-local counter
        in[rank] += 1 - int(x * x + y * y); // no conditional version (faster)
    }
}

int main(int argc, char* argv[])
{
    // querying device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    cuda_error_check();

    // set the number of threads
    size_t n_threads = prop.maxThreadsPerBlock;
    if (3 == argc)
    {
        int n = atoi(argv[2]);
        if (0 == n)
        {
            cout << "Usage: "
                 << argv[0]
                 << " [number of tosses] [number of threads]"
                 << endl;
            return -1;
        }
        if (n < n_threads)
            n_threads = n;
    }

    // read total number of tosses from the command line
    size_t n_tosses = process_cmdline(argc, argv);
    if (0 == n_tosses)
        return -1;

    cout << "Monte-Carlo Pi Estimator\n"
         << "Method: CUDA (GPU) -- "
         << n_threads << " thread(s)\n"
         << "Device name: " << prop.name
         << "\nNumber of tosses: " << n_tosses << endl;

    // run the simulation and time it...
    //------> start timer
    timer::time_point start = timer::now();

    // memory for thread local results
    size_t* in_device;
    hipMalloc(&in_device, n_threads * sizeof(size_t));
    cuda_error_check();
    // start parallel Monte Carlo
    cuda_toss<<<1, n_threads>>>(n_tosses, in_device);
    cuda_error_check();

    // reducing...
    vector<size_t> in(n_threads);
    hipMemcpy( 
        in.data()
    ,   in_device
    ,   n_threads * sizeof(size_t)
    ,   hipMemcpyDeviceToHost);
    cuda_error_check();
    hipFree(in_device);
    size_t n_in_circle{0};
    for (size_t i{0}; i < n_threads; ++i)
        n_in_circle += in[i];

    timer::duration elapsed = timer::now() - start;
    //------> end timer

    // ouput the results
    const long double pi = 3.141592653589793238462643L; // 25-digit Pi
    long double pi_estimate = 4.0L * n_in_circle / n_tosses;
    cout << "Estimated Pi: " << fixed << setw(17) << setprecision(15)
         << pi_estimate << endl
         << "Percent error: " << setprecision(3)
         << abs(pi_estimate - pi) / pi * 100.0 << '%' << endl
         << "Elapsed time: "
         << chrono::duration_cast<chrono::milliseconds>(elapsed).count()
         << " ms" << endl;
}
